#include "hip/hip_runtime.h"
#include <stdio.h>
#include<Windows.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
using namespace std;
#define N 1024
#define TILE_WIDTH 16

void print_matrix(double* matrix) {
    cout << endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << " " << matrix[i * N + j];
        }
        cout << endl;
    }
}

__global__ void matrixMulGPU(double* a, double* b, double* c)
{
    int val = 0;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < N)
    {
        for (int k = 0; k < N; ++k)
            val += a[row * N + k] * b[k * N + col];
        c[row * N + col] = val;
    }
}


__global__ void matrixMulGPU_pro(double* a, double* b, double* c)
{
    __shared__ double Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ double Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Col = bx * TILE_WIDTH + tx;
    int Row = by * TILE_WIDTH + ty;

    int val = 0;

    for (int i = 0; i < N / TILE_WIDTH; i++)  //�ж��ٸ�TILE_WIDTH��ÿ��ѭ������һ����Ĵ�С
    {
        Mds[ty][tx] = a[Row * N + (i * TILE_WIDTH + tx)];
        Nds[ty][tx] = b[Col + (i * TILE_WIDTH + ty) * N];
        __syncthreads();


        for (int k = 0; k < TILE_WIDTH; k++) //TILE_WIDTH���
            val += Mds[ty][k] * Nds[k][tx];
        __syncthreads();
    }

    c[Row * N + Col] = val;
}


void cublas_matrixMul()
{
    double* h_A = (double*)malloc(sizeof(double) * N * N);
    double* h_B = (double*)malloc(sizeof(double) * N * N);
    double* h_C = (double*)malloc(sizeof(double) * N * N);
    for (int i = 0; i < N * N; i++) {
        h_A[i] = i;
        h_B[i] = i;
    }

    double* g_A, * g_B, * g_C_cubals;
    hipMalloc((void**)&g_A, sizeof(double) * N * N);
    hipMalloc((void**)&g_B, sizeof(double) * N * N);
    hipMalloc((void**)&g_C_cubals, sizeof(double) * N * N);

    hipblasSetVector(N * N, sizeof(double), h_A, 1, g_A, 1);
    hipblasSetVector(N * N, sizeof(double), h_B, 1, g_B, 1);
    hipblasSetVector(N * N, sizeof(double), h_C, 1, g_C_cubals, 1);

    hipblasHandle_t  handle;
    hipblasCreate(&handle);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, g_A, N, g_B, N, &beta, g_C_cubals, N);
    hipblasDestroy(handle);

    //hipblasGetVector(N * N, sizeof(double), g_C_cubals, 1, h_C, 1);
    //print_matrix(h_C);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(g_A);
    hipFree(g_B);
    hipFree(g_C_cubals);

}

void cal_sum_test()
{
    cout << endl;
    LARGE_INTEGER nFreq;
    LARGE_INTEGER nBeginTime;
    LARGE_INTEGER nEndTime;

    QueryPerformanceFrequency(&nFreq);
    double* h_A = (double*)malloc(sizeof(double) * N * N);
    double* h_B = (double*)malloc(sizeof(double) * N * N);
    double* h_C = (double*)malloc(sizeof(double) * N * N);
    for (int i = 0; i < N * N; i++) {
        h_A[i] = i;
        h_B[i] = i;
    }

    double* g_A, * g_B, * g_C_cubals,*g_C_my;
    hipMalloc((void**)&g_A, sizeof(double) * N * N);
    hipMalloc((void**)&g_B, sizeof(double) * N * N);
    hipMalloc((void**)&g_C_cubals, sizeof(double) * N * N);
    hipMalloc((void**)&g_C_my, sizeof(double) * N * N);

    hipblasSetVector(N * N, sizeof(double), h_A, 1, g_A, 1);
    hipblasSetVector(N * N, sizeof(double), h_B, 1, g_B, 1);
    hipblasSetVector(N * N, sizeof(double), h_C, 1, g_C_cubals, 1);
    hipblasSetVector(N * N, sizeof(double), h_C, 1, g_C_my, 1);

    QueryPerformanceCounter(&nBeginTime);

    hipblasHandle_t  handle;
    hipblasCreate(&handle);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, g_A, N, g_B, N, &beta, g_C_cubals, N);
    hipblasDestroy(handle);


    QueryPerformanceCounter(&nEndTime);
    cout << "    cublas_matrix_multiply " << (double)(nEndTime.QuadPart - nBeginTime.QuadPart) * 1.0 / nFreq.QuadPart << endl;


    QueryPerformanceCounter(&nBeginTime);

    dim3 threads_per_block_(16, 16, 1);
    dim3 number_of_blocks_((N / threads_per_block_.x) + 1, (N / threads_per_block_.y) + 1, 1);


    matrixMulGPU << < number_of_blocks_, threads_per_block_ >> > (g_A, g_B, g_C_my);

    hipDeviceSynchronize();
    QueryPerformanceCounter(&nEndTime);
    cout << "    my_matrix_multiply     " << (double)(nEndTime.QuadPart - nBeginTime.QuadPart) * 1.0 / nFreq.QuadPart << endl;

    QueryPerformanceCounter(&nBeginTime);

    dim3 threads_per_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 number_of_blocks(N / TILE_WIDTH, N / TILE_WIDTH, 1);
    

    matrixMulGPU_pro<< < number_of_blocks, threads_per_block >> > (g_A, g_B, g_C_my);

    hipDeviceSynchronize();
    QueryPerformanceCounter(&nEndTime);
    cout<<"    my_matrix_multiply_pro "<< (double)(nEndTime.QuadPart - nBeginTime.QuadPart)*1.0/nFreq.QuadPart<<endl;


    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(g_A);
    hipFree(g_B);
    hipFree(g_C_cubals);
    hipFree(g_C_my);
}






int main()
{
    cal_sum_test();
    return 0;
}
