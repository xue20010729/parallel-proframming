#include <stdio.h>
#include<Windows.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
using namespace std;
#define N 3

//void cal_sum_test()
//{
//    LARGE_INTEGER nFreq;
//    LARGE_INTEGER nBeginTime;
//    LARGE_INTEGER nEndTime;
//
//    QueryPerformanceFrequency(&nFreq);
//
//
//    QueryPerformanceCounter(&nEndTime);
//    cout<<" GPU time (global memory):"<< (double)(nEndTime.QuadPart - nBeginTime.QuadPart)*1.0/nFreq.QuadPart<<endl;
//
//
//}

void print_matrix(double *matrix){
    cout << endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout <<" " << matrix[i * N + j] ;
        }
        cout << endl;
    }
}
void test()
{
    double* h_A = (double*)malloc(sizeof(double) * N * N);
    double* h_B = (double*)malloc(sizeof(double) * N * N);
    double* h_C = (double*)malloc(sizeof(double) * N * N);
    for (int i = 0; i < N * N; i++) {
        h_A[i] = i;
        h_B[i] = i;
    }

    double* g_A, * g_B, * g_C;
    hipMalloc((void**)&g_A, sizeof(double) * N * N);
    hipMalloc((void**)&g_B, sizeof(double) * N * N);
    hipMalloc((void**)&g_C, sizeof(double) * N * N);

    hipblasSetVector(N * N, sizeof(double), h_A, 1, g_A, 1);
    hipblasSetVector(N * N, sizeof(double), h_B, 1, g_B, 1);
    hipblasSetVector(N * N, sizeof(double), h_C, 1, g_C, 1);

    hipblasHandle_t  handle;
    hipblasCreate(&handle);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, g_A, N, g_B, N, &beta, g_C, N);
    hipblasDestroy(handle);

    hipblasGetVector(N * N, sizeof(double), g_C, 1, h_C, 1);
    print_matrix(h_C);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(g_A);
    hipFree(g_B);
    hipFree(g_C);

}


int main()
{
    test();
    return 0;
}
